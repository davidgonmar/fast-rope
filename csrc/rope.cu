#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_complex.h>

constexpr size_t THREADBLOCK_SIZE = 1024;
constexpr size_t MAX_BLOCKS_X = 65535;
constexpr size_t MAX_BLOCKS_Y = 65535;



__global__ void rope_kernel(
    const float* __restrict__ sequence,
    const float* __restrict__ freqs,
    float* __restrict__ output,
    const size_t batch_size,
    const size_t seq_len,
    const size_t d_model
) {
    // input and output are both of shape (batch_size, seq_len, d_model)
    // freqs is of shape (seq_len, d_model)

    for (size_t batch_idx = blockIdx.x; batch_idx < batch_size; batch_idx += gridDim.x) {
        for (size_t seq_idx = blockIdx.y; seq_idx < seq_len; seq_idx += gridDim.y) {
            // d_idx 2 by 2
            for (size_t d_idx = threadIdx.x * 2; d_idx < d_model; d_idx += blockDim.x * 2) {
                // first, load d_idx and d_idx + 2 from sequence
                const float x1 = sequence[batch_idx * seq_len * d_model + seq_idx * d_model + d_idx];
                const float x2 = sequence[batch_idx * seq_len * d_model + seq_idx * d_model + d_idx + 1];
                // now handle x as a complex number
                const hipFloatComplex x = make_hipFloatComplex(x1, x2);
                // load the corresponding frequency
                const float f1 = freqs[(seq_idx * d_model / 2) + d_idx / 2];
                const hipFloatComplex f = make_hipFloatComplex(cosf(f1), sinf(f1)); // into polar form
                // we need to exp the frequency
                const hipFloatComplex x_rot = hipCmulf(x, f);

                output[batch_idx * seq_len * d_model + seq_idx * d_model + d_idx] = hipCrealf(x_rot);
                output[batch_idx * seq_len * d_model + seq_idx * d_model + d_idx + 1] = hipCimagf(x_rot);
            }
        }
    } 
}



torch::Tensor rope_forward(
    const torch::Tensor& sequence,
    const torch::Tensor& freqs
) { 
    torch::Tensor output = torch::empty_like(sequence);
    const size_t batch_size = sequence.size(0);
    const size_t seq_len = sequence.size(1);
    const size_t d_model = sequence.size(2);

    TORCH_CHECK(d_model % 2 == 0, "d_model must be even");


    // Check sequence and freqs have the same shape except batch_size
    TORCH_CHECK(sequence.size(1) == freqs.size(0), "sequence and freqs must have the same shape seq_len");
    TORCH_CHECK(sequence.size(2) == freqs.size(1) * 2, "sequence must have double the size of freqs in the last dimension");


    const dim3 grid_size(
        std::min(MAX_BLOCKS_X, batch_size),
        std::min(MAX_BLOCKS_Y, seq_len)
    );
    const dim3 block_size(THREADBLOCK_SIZE);

    rope_kernel<<<grid_size, block_size>>>(
        sequence.data_ptr<float>(),
        freqs.data_ptr<float>(),
        output.data_ptr<float>(),
        batch_size,
        seq_len,
        d_model
    );
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error("Error in rope_forward: " + std::string(hipGetErrorString(err)));
    }

    return output;
}


PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("rope_forward", &rope_forward, "Rope forward (CUDA)");
}